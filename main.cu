
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdint>
#include <cstring>
#include <iostream>
#include <limits>

constexpr uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

constexpr int block_size_u32 = 512 / 32;
constexpr int hash_size_u32  = 256 / 32;
constexpr int block_size_u8  = 512 / 8;
constexpr int num_inputs_u8  = block_size_u8 - 8 - 4; // len in u64 and bit padding in u32
constexpr int num_inputs_u32 = num_inputs_u8 / 4;

using block_t = uint32_t[block_size_u32]; // hash algorithm input
using hash_t  = uint32_t[hash_size_u32]; // hash algorithm output, each byte is a hexadecimal value
/// \brief Represents one big 128-bits unsigned integer, lower is a better score.
using score_t = uint32_t[hash_size_u32];

uint32_t rotr(uint32_t a, int b) { return (a >> b) | (a << (32 - b)); }
uint32_t ch(uint32_t x, uint32_t y, uint32_t z) { return (x & y) ^ (~x & z); }
uint32_t maj(uint32_t x, uint32_t y, uint32_t z) { return (x & y) ^ (x & z) ^ (y & z); }
uint32_t ep0(uint32_t x) { return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22); }
uint32_t ep1(uint32_t x) { return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25); }
uint32_t sig0(uint32_t x) { return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3); }
uint32_t sig1(uint32_t x) { return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10); }

uint32_t swap_endian(uint32_t x)
{
    const uint8_t* ptr = reinterpret_cast<const uint8_t*>(&x);
    return uint32_t{ptr[3]} | (uint32_t{ptr[2]} << 8) | (uint32_t{ptr[1]} << 16) |
           (uint32_t{ptr[0]} << 24);
}

void sha256(hash_t& hash, const block_t& block)
{
    uint32_t m[64];
    for (int i = 0; i < block_size_u32; ++i) {
        m[i] = block[i];
    }
    for (int i = 16; i < 64; ++i) {
        m[i] = sig1(m[i - 2]) + m[i - 7] + sig0(m[i - 15]) + m[i - 16];
    }

    hash[0] = 0x6a09e667;
    hash[1] = 0xbb67ae85;
    hash[2] = 0x3c6ef372;
    hash[3] = 0xa54ff53a;
    hash[4] = 0x510e527f;
    hash[5] = 0x9b05688c;
    hash[6] = 0x1f83d9ab;
    hash[7] = 0x5be0cd19;

    uint32_t a = hash[0];
    uint32_t b = hash[1];
    uint32_t c = hash[2];
    uint32_t d = hash[3];
    uint32_t e = hash[4];
    uint32_t f = hash[5];
    uint32_t g = hash[6];
    uint32_t h = hash[7];

    for (int i = 0; i < 64; ++i) {
        const uint32_t t1 = h + ep1(e) + ch(e, f, g) + k[i] + m[i];
        const uint32_t t2 = ep0(a) + maj(a, b, c);

        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    hash[0] = swap_endian(hash[0] + a);
    hash[1] = swap_endian(hash[1] + b);
    hash[2] = swap_endian(hash[2] + c);
    hash[3] = swap_endian(hash[3] + d);
    hash[4] = swap_endian(hash[4] + e);
    hash[5] = swap_endian(hash[5] + f);
    hash[6] = swap_endian(hash[6] + g);
    hash[7] = swap_endian(hash[7] + h);
}

void print_input(const block_t& block)
{
    for (int i = 0; i < num_inputs_u32; ++i) {
        const uint32_t tmp = swap_endian(block[i]);
        for (int j = 0; j < 4; ++j) {
            printf("%c", reinterpret_cast<const char*>(&tmp)[j]);
        }
    }
    printf("\n");
}

void print_hash(const hash_t& hash)
{
    for (int i = 0; i < 8; ++i) {
        printf("%08x", swap_endian(hash[i]));
    }
    printf("\n");
}

void score_hash(score_t& score, const hash_t& hash)
{
    for (int i = 0; i < 4; ++i) {
        uint32_t tmp{};
        for (int j = 0; j < 8; ++j) {
            const uint8_t val = reinterpret_cast<const uint8_t*>(hash)[8 * i + j];
            // for (int k = 0; k < 2; ++k)
            // {
            //     const uint8_t hex = val >> (k - i - 1) * 4;
            // [[maybe_unused]] const bool is_digit            = 48 <= hex && hex <= 57;
            // [[maybe_unused]] const bool is_lowercase_xdigit = 97 <= hex && hex <= 102;
            // assert(is_digit || is_lowercase_xdigit);
            // const uint8_t val = hex < 58 ? hex - 48 : hex - 97 + 10;
            // assert(val < 16);
            tmp |= val << (8 - j - 1) * 4;
            // }
        }
        score[i] = tmp;
    }
}

bool less_than(const hash_t& lhs, const hash_t& rhs)
{
    for (int i = 0; i < 4; ++i) {
        const uint32_t lhs_u32 = swap_endian(lhs[i]);
        const uint32_t rhs_u32 = swap_endian(rhs[i]);
        if (lhs_u32 < rhs_u32) {
            return true;
        } else if (rhs_u32 < lhs_u32) {
            return false;
        }
    }
    return false;
}

uint8_t base64_to_ascii(int x)
{
    return x < 26 ? 65 + x : x < 52 ? 71 + x : x < 62 ? x - 4 : x < 63 ? 43 : 47;
}

int main()
{
    uint32_t block[block_size_u32];
    block[0] = uint32_t{'n'} << 24 | (uint32_t{'o'} << 16) | (uint32_t{'l'} << 8) | (uint32_t{'/'});
    for (int i = 0; i < num_inputs_u32; ++i) {
        block[1 + i] =
            uint32_t{'0'} | (uint32_t{'0'} << 8) | (uint32_t{'0'} << 16) | (uint32_t{'0'} << 24);
    }
    block[block_size_u32 - 3] = swap_endian(uint32_t{0x80}); // single bit padding
    block[block_size_u32 - 2] = 0;
    // length, 64 - 8 - 4 = 52 * 8 = 416 in u32 big endian
    block[block_size_u32 - 1] = uint32_t{416};

    hash_t best_hash{};
    std::fill(best_hash, best_hash + 8, std::numeric_limits<uint32_t>::max());

    for (int i = 0; i < 64; ++i) {
        const uint32_t mask_i = base64_to_ascii(i) << 24;
        for (int j = 0; j < 64; ++j) {
            const uint32_t mask_j = base64_to_ascii(j) << 16;
            for (int k = 0; k < 64; ++k) {
                const uint32_t mask_k = base64_to_ascii(k) << 8;
                for (int l = 0; l < 64; ++l) {
                    const uint32_t mask_l     = base64_to_ascii(k);
                    block[num_inputs_u32 - 1] = mask_i | mask_j | mask_k | mask_l;

                    uint32_t hash[hash_size_u32];
                    sha256(hash, block);

                    if (less_than(hash, best_hash)) {
                        print_input(block);
                        std::memcpy(best_hash, hash, sizeof(hash));
                        print_hash(hash);
                    }
                }
            }
        }
    }
}
